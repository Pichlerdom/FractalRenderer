#include "hip/hip_runtime.h"
#include "fractal.h"

CudaFractalGenerator::CudaFractalGenerator(uint32_t w, uint32_t h){
  m_w = w;
  m_h = h;
  
  hipMalloc((void **) &d_pixel_buffer,
	     sizeof(uint8_t) * m_w * m_h * 4);

}

CudaFractalGenerator::~CudaFractalGenerator(){

  hipFree(d_pixel_buffer);

}

void CudaFractalGenerator::generate_fractal(uint8_t *pixel_buffer,
					    double world_x, double world_y,
					    double world_width, double world_height,
					    uint32_t max_iterations){

  dim3 block(BLOCK_N, BLOCK_N);
  dim3 grid((uint32_t) ceil( (double)m_w / (double)BLOCK_N ),
	    (uint32_t) ceil( (double)m_h / (double)BLOCK_N ));

  fractal_kernel<<<grid,block>>>(d_pixel_buffer,
				 m_w, m_h, 
				 world_x, world_y,
				 world_width, world_height,
				 max_iterations);

  hipMemcpy((void*) pixel_buffer,
	     (void*) d_pixel_buffer,
	     sizeof(uint8_t) * m_w * m_h * 4,
	     hipMemcpyDeviceToHost);
  
}
